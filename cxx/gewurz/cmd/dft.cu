#include "hip/hip_runtime.h"
#define LIBCUDACXX_ENABLE_SIMPLIFIED_COMPLEX_OPERATIONS
#include "rl/io/hd5.hpp"
#include "rl/log/log.hpp"

#include "../algo/lsmr.cuh"
#include "../args.hpp"
#include "../op/dft.cuh"
#include "../op/recon.cuh"
#include "../sense.hpp"
#include "info.hpp"

#include <thrust/extrema.h>

using namespace rl;

auto ReadTrajectory(HD5::Reader &reader) -> DTensor<TDev, 3>
{
  Log::Print("gewurz", "Read trajectory");
  auto const shape = reader.dimensions("trajectory");
  auto const nS = shape[1];
  auto const nT = shape[2];

  HTensor<float, 3> hT(3L, nS, nT);
  reader.readTo(hT.vec.data(), HD5::Keys::Trajectory);
  HTensor<TDev, 3> hhT(3L, nS, nT);
  thrust::transform(hT.vec.begin(), hT.vec.end(), hhT.vec.begin(), ConvertTo);
  DTensor<TDev, 3> T(3L, nS, nT);
  thrust::copy(hhT.vec.begin(), hhT.vec.end(), T.vec.begin());
  return T;
}

void WriteTrajectory(DTensor<TDev, 3> const &T, HD5::Shape<3> const mat, HD5::Writer &writer)
{
  Log::Print("gewurz", "Write trajectory");
  auto const nS = T.span.extent(1);
  auto const nT = T.span.extent(2);

  HTensor<TDev, 3> hhT(3L, nS, nT);
  thrust::copy(T.vec.begin(), T.vec.end(), hhT.vec.begin());
  HTensor<float, 3> hT(3L, nS, nT);
  thrust::transform(hhT.vec.begin(), hhT.vec.end(), hT.vec.begin(), ConvertFrom);

  writer.writeTensor(HD5::Keys::Trajectory, HD5::Shape<3>(3L, nS, nT), hT.vec.data(), HD5::Dims::Trajectory);
  writer.writeAttribute(HD5::Keys::Trajectory, "matrix", mat);
}

void DoTest(DTensor<TDev, 3> const &T, rl::HD5::Shape<3> const mat)
{
  Log::Print("gewurz", "Test");
  auto const             nS = T.span.extent(1);
  auto const             nT = T.span.extent(2);
  UTensor<CuCx<TDev>, 2> Mks(nS, nT);
  UTensor<CuCx<TDev>, 3> Mimg(mat[0], mat[1], mat[2]);

  thrust::fill(Mks.vec.begin(), Mks.vec.end(), CuCx<TDev>(1));
  // for (int it = 0; it < nT; it++) {
  //   Mks.span(nS - 1, it) = CuCx<TDev>(1);
  // }

  HD5::Writer                     debug("test.h5");
  HTensor<CuCx<TDev>, 2>          hks(nS, nT);
  HTensor<std::complex<float>, 2> hhks(nS, nT);
  thrust::copy(Mks.vec.begin(), Mks.vec.end(), hks.vec.begin());
  thrust::transform(hks.vec.begin(), hks.vec.end(), hhks.vec.begin(), ToStdCx);
  debug.writeTensor("Mks1", HD5::Shape<2>{nS, nT}, hhks.vec.data(), {"s", "t"});

  gw::DFT::ThreeD dft{T.span};
  dft.adjoint(Mks.span, Mimg.span);
  Log::Print("Test", "|img| {} |ks| {}", gw::CuNorm(Mimg.vec), gw::CuNorm(Mks.vec));

  HTensor<CuCx<TDev>, 3>          himg(mat[0], mat[1], mat[2]);
  HTensor<std::complex<float>, 3> hhimg(mat[0], mat[1], mat[2]);
  thrust::copy(Mimg.vec.begin(), Mimg.vec.end(), himg.vec.begin());
  thrust::transform(himg.vec.begin(), himg.vec.end(), hhimg.vec.begin(), ToStdCx);
  debug.writeTensor("Mimg", HD5::Shape<3>{mat[0], mat[1], mat[2]}, hhimg.vec.data(), {"i", "j", "k"});

  dft.forward(Mimg.span, Mks.span);
  Log::Print("Test", "|img| {} |ks| {}", gw::CuNorm(Mimg.vec), gw::CuNorm(Mks.vec));

  thrust::copy(Mks.vec.begin(), Mks.vec.end(), hks.vec.begin());
  thrust::transform(hks.vec.begin(), hks.vec.end(), hhks.vec.begin(), ToStdCx);
  debug.writeTensor("Mks2", HD5::Shape<2>{nS, nT}, hhks.vec.data(), {"s", "t"});
}

auto Preconditioner(DTensor<TDev, 3> const &T, int const nI, int const nJ, int const nK) -> DTensor<TDev, 2>
{
  Log::Print("gewurz", "Preconditioner");
  auto const       nS = T.span.extent(1);
  auto const       nT = T.span.extent(2);
  DTensor<TDev, 2> M(nS, nT);
  thrust::fill(M.vec.begin(), M.vec.end(), TDev(1));

  DTensor<CuCx<TDev>, 2> Mks(nS, nT);
  DTensor<CuCx<TDev>, 3> Mimg(nI, nJ, nK);
  thrust::fill(Mks.vec.begin(), Mks.vec.end(), CuCx<TDev>(1));

  HD5::Writer debug("debug.h5");

  gw::DFT::ThreeD dft{T.span};
  Log::Print("Precon", "|img| {} |ks| {}", gw::CuNorm(Mimg.vec), gw::CuNorm(Mks.vec));
  dft.adjoint(Mks.span, Mimg.span);
  Log::Print("Precon", "|img| {} |ks| {}", gw::CuNorm(Mimg.vec), gw::CuNorm(Mks.vec));
  dft.forward(Mimg.span, Mks.span);
  Log::Print("Precon", "|img| {} |ks| {}", gw::CuNorm(Mimg.vec), gw::CuNorm(Mks.vec));
  float const λ = 0.0f;
  thrust::transform(thrust::cuda::par, Mks.vec.begin(), Mks.vec.end(), M.vec.begin(),
                    [λ] __device__(CuCx<TDev> x) { return TDev(1 + λ) / (cuda::std::abs(x) + λ); });
  auto const mm = thrust::minmax_element(thrust::cuda::par, M.vec.begin(), M.vec.end());
  TDev const min = *(mm.first);
  TDev const max = *(mm.second);
  Log::Print("Precon", "|M| {} Min {} Max {}", gw::CuNorm(M.vec), min, max);
  return M;
}

auto ReadImage(rl::HD5::Reader &reader) -> DTensor<CuCx<TDev>, 4>
{
  Log::Print("gewurz", "Read k-space");
  auto const shape = reader.dimensions();
  auto const nC = shape[3];
  auto const nI = shape[0];
  auto const nJ = shape[1];
  auto const nK = shape[2];
  fmt::print(stderr, "{} {} {} {}\n", nI, nJ, nK, nC);
  HTensor<std::complex<float>, 4> himg(nI, nJ, nK, nC);
  reader.readTo(himg.vec.data());
  fmt::print(stderr, "|himg| {}\n", gw::CuNorm(himg.vec));
  HTensor<CuCx<TDev>, 4> hhimg(nI, nJ, nK, nC);
  std::transform(himg.vec.begin(), himg.vec.end(), hhimg.vec.begin(), FromStdCx);
  fmt::print(stderr, "|hhimg| {}\n", gw::CuNorm(hhimg.vec));
  DTensor<CuCx<TDev>, 4> img(nI, nJ, nK, nC);
  thrust::copy(hhimg.vec.begin(), hhimg.vec.end(), img.vec.begin());
  fmt::print(stderr, "|img| {}\n", gw::CuNorm(img.vec));
  return img;
}

auto ReadKS(rl::HD5::Reader &reader) -> DTensor<CuCx<TDev>, 3>
{
  Log::Print("gewurz", "Read k-space");
  auto const                      shape = reader.dimensions();
  auto const                      nC = shape[0];
  auto const                      nS = shape[1];
  auto const                      nT = shape[2];
  HTensor<std::complex<float>, 3> hKS(nC, nS, nT);
  reader.readTo(hKS.vec.data());
  HTensor<CuCx<TDev>, 3> hhKS(nC, nS, nT);
  std::transform(hKS.vec.begin(), hKS.vec.end(), hhKS.vec.begin(), FromStdCx);
  DTensor<CuCx<TDev>, 3> ks(nC, nS, nT);
  thrust::copy(hhKS.vec.begin(), hhKS.vec.end(), ks.vec.begin());
  return ks;
}

auto ReadSENSE(std::string const &sname, rl::HD5::Shape<3> const mat, int const nC) -> DTensor<CuCx<TDev>, 4>
{
  auto                   hostS = gw::GetSENSE(sname, mat);
  HTensor<CuCx<TDev>, 4> hhS(mat[0], mat[1], mat[2], nC);
  DTensor<CuCx<TDev>, 4> S(mat[0], mat[1], mat[2], nC);
  std::transform(hostS.begin(), hostS.end(), hhS.vec.begin(), FromStdCx);
  thrust::copy(hhS.vec.begin(), hhS.vec.end(), S.vec.begin());
  return S;
}

template <int NC> void DoRecon(DTensor<CuCx<TDev>, 3> const    &ks,
                               DTensor<TDev, 3> const          &T,
                               DTensor<TDev, 2> const          &M,
                               DTensor<CuCx<TDev>, 4> const    &S,
                               HTensor<std::complex<float>, 3> &hImg)

{
  Log::Print("gewurz", "Recon");
  auto const    nI = hImg.span.extent(0);
  auto const    nJ = hImg.span.extent(1);
  auto const    nK = hImg.span.extent(2);
  auto          Minv = gw::MulPacked<CuCx<TDev>, TDev, 3>{M.span};
  gw::Recon<NC> A{S.span, T.span};

  DTensor<CuCx<TDev>, 3>     img(nI, nJ, nK);
  gw::LSMR<CuCx<TDev>, 3, 3> lsmr{&A, &Minv};
  lsmr.run(ks, img);
  HTensor<CuCx<TDev>, 3> hhImg(nI, nJ, nK);
  thrust::copy(img.vec.begin(), img.vec.end(), hhImg.vec.begin());
  thrust::transform(hhImg.vec.begin(), hhImg.vec.end(), hImg.vec.begin(), ToStdCx);
}

template <int NC>
void DoForwardDFT(DTensor<CuCx<TDev>, 4> const &imgs, DTensor<TDev, 3> const &T, HTensor<std::complex<float>, 3> &hKS)
{
  Log::Print("gewurz", "Recon");
  auto const                nC = imgs.span.extent(3);
  auto const                nI = imgs.span.extent(0);
  auto const                nJ = imgs.span.extent(1);
  auto const                nK = imgs.span.extent(2);
  auto const                nS = hKS.span.extent(1);
  auto const                nT = hKS.span.extent(2);
  gw::DFT::ThreeDPacked<NC> A{T.span};
  DTensor<CuCx<TDev>, 3>    ks(nC, nS, nT);
  A.forward(imgs.span, ks.span);
  fmt::print(stderr, "|img| {} |ks| {}\n", gw::CuNorm(imgs.vec), gw::CuNorm(ks.vec));
  HTensor<CuCx<TDev>, 3> hhKS(nC, nS, nT);
  thrust::copy(ks.vec.begin(), ks.vec.end(), hhKS.vec.begin());
  thrust::transform(hhKS.vec.begin(), hhKS.vec.end(), hKS.vec.begin(), ToStdCx);
  fmt::print(stderr, "|hKs| {} |hhKS| {}\n", gw::CuNorm(hKS.vec), gw::CuNorm(hhKS.vec));
}

template <int NC>
void DoAdjointDFT(DTensor<CuCx<TDev>, 3> const &ks, DTensor<TDev, 3> const &T, HTensor<std::complex<float>, 4> &hImg)
{
  Log::Print("gewurz", "Recon");
  auto const nC = hImg.span.extent(3);
  auto const nI = hImg.span.extent(0);
  auto const nJ = hImg.span.extent(1);
  auto const nK = hImg.span.extent(2);

  gw::DFT::ThreeDPacked<NC> A{T.span};
  DTensor<CuCx<TDev>, 4>    imgs(nI, nJ, nK, nC);
  A.adjoint(ks.span, imgs.span);

  HTensor<CuCx<TDev>, 4> hhImg(nI, nJ, nK, nC);
  thrust::copy(imgs.vec.begin(), imgs.vec.end(), hhImg.vec.begin());
  thrust::transform(hhImg.vec.begin(), hhImg.vec.end(), hImg.vec.begin(), FromStdCx);
}

template <int NC> void DoInverseDFT(DTensor<CuCx<TDev>, 3> const    &ks,
                                    DTensor<TDev, 3> const          &T,
                                    bool const                       precon,
                                    HTensor<std::complex<float>, 4> &hImg)
{
  Log::Print("gewurz", "Recon");
  auto const nC = hImg.span.extent(3);
  auto const nI = hImg.span.extent(0);
  auto const nJ = hImg.span.extent(1);
  auto const nK = hImg.span.extent(2);

  DTensor<CuCx<TDev>, 4> imgs(nI, nJ, nK, nC);
  gw::DFT::ThreeDPacked<NC>          A{T.span};
  
  if (precon) {
    auto const                         W = Preconditioner(T, nI, nJ, nK);
    gw::MulPacked<CuCx<TDev>, TDev, 3> Minv{W.span};
    gw::LSMR lsmr{&A, &Minv};
    lsmr.run(ks, imgs);
  } else {
    gw::LSMR lsmr{&A};
    lsmr.run(ks, imgs);
  }

  HTensor<CuCx<TDev>, 4> hhImg(nI, nJ, nK, nC);
  thrust::copy(imgs.vec.begin(), imgs.vec.end(), hhImg.vec.begin());
  thrust::transform(hhImg.vec.begin(), hhImg.vec.end(), hImg.vec.begin(), FromStdCx);
}

void main_dft(args::Subparser &parser)
{
  args::Positional<std::string> iname(parser, "FILE", "Input HD5 file");
  args::Positional<std::string> oname(parser, "FILE", "Output HD5 file");
  args::Flag                    adj(parser, "A", "Adjoint only", {'a', "adj"});
  args::Flag                    fwd(parser, "F", "Forward", {'f', "fwd"});
  args::Flag                    noM(parser, "M", "No preconditioning", {'p', "nop"});

  ParseCommand(parser, iname, oname);
  Log::Print("DFT", "Welcome!");

  HD5::Reader reader(iname.Get());
  auto const  mat = reader.readAttributeShape<3>(HD5::Keys::Trajectory, "matrix");
  auto const  T = ReadTrajectory(reader);
  auto const  info = reader.readStruct<gw::Info>(HD5::Keys::Info);
  HD5::Writer writer(oname.Get());
  WriteTrajectory(T, mat, writer);
  writer.writeStruct(HD5::Keys::Info, info);

  if (fwd) {
    auto const shape = reader.dimensions();
    auto const nC = shape[3];
    auto const nS = T.span.extent(1);
    auto const nT = T.span.extent(2);
    auto       img = ReadImage(reader);
    fmt::print(stderr, "|img| {}\n", gw::CuNorm(img.vec));
    HTensor<std::complex<float>, 3> ks(nC, nS, nT);
    switch (nC) {
    case 1: DoForwardDFT<1>(img, T, ks); break;
    case 2: DoForwardDFT<2>(img, T, ks); break;
    case 4: DoForwardDFT<4>(img, T, ks); break;
    case 8: DoForwardDFT<8>(img, T, ks); break;
    default: throw(Log::Failure("DFT", "Unsupported number of channels {}", nC));
    }
    fmt::print(stderr, "|ks| {}\n", gw::CuNorm(ks.vec));
    writer.writeTensor("data", HD5::Shape<5>{nC, nS, nT, 1, 1}, ks.vec.data(), {"channel", "sample", "trace", "slab", "t"});
  } else {
    auto const                      shape = reader.dimensions();
    auto const                      nC = shape[0];
    auto                            KS = ReadKS(reader);
    HTensor<std::complex<float>, 4> img(mat[0], mat[1], mat[2], nC);
    if (adj) {
      switch (nC) {
      case 1: DoAdjointDFT<1>(KS, T, img); break;
      case 2: DoAdjointDFT<2>(KS, T, img); break;
      case 4: DoAdjointDFT<4>(KS, T, img); break;
      case 8: DoAdjointDFT<8>(KS, T, img); break;
      default: throw(Log::Failure("DFT", "Unsupported number of channels {}", nC));
      }
    } else {
      switch (nC) {
      case 1: DoInverseDFT<1>(KS, T, !noM, img); break;
      case 2: DoInverseDFT<2>(KS, T, !noM, img); break;
      case 4: DoInverseDFT<4>(KS, T, !noM, img); break;
      case 8: DoInverseDFT<8>(KS, T, !noM, img); break;
      default: throw(Log::Failure("DFT", "Unsupported number of channels {}", nC));
      }
    }
    writer.writeTensor("data", HD5::Shape<4>{mat[0], mat[1], mat[2], nC}, img.vec.data(), {"i", "j", "k", "channel"});
  }

  Log::Print("DFT", "Finished");
}
